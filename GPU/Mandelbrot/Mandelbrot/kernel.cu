#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include "mandel.h"
#include "mandel_constants_gpu.h"

__global__ void mandel_kernel(pfc::byte_t* dest, int const size, int const o) {
	auto const i{ blockIdx.x * blockDim.x + threadIdx.x };
	if (i < size) {
		dest[i] = colorDevice((i % WIDTH) * X_FACTORS[o] + CX_MIN[o], (i / WIDTH) * Y_FACTORS[o] + CY_MIN[o]);
	}
}

__global__ void mandel_kernel(pfc::BGR_4_t* dest, int const size, int const o) {
	auto const i{ blockIdx.x * blockDim.x + threadIdx.x };

	if (i < size) { // Unnecessary Problem size can be devided by 32 or 64, whatever group size the funciton is called with
		auto const x{ i % WIDTH };
		auto const y{ i / WIDTH };
		dest[i].red = std::move(colorDevice(x * X_FACTORS[o] + CX_MIN[o], y * Y_FACTORS[o] + CY_MIN[o]));
	}
}

void call_mandel_kernel(dim3 const& big, dim3 const& tib, pfc::byte_t* dest, int const size, int const outer_idx) {
	mandel_kernel << <big, tib >> > (dest, size, outer_idx);
}

void call_mandel_kernel(dim3 const& big, dim3 const& tib, pfc::BGR_4_t* dest, int const size, int const outer_idx) {

	mandel_kernel << <big, tib >> > (dest, size, outer_idx);
}

void call_mandel_kernel(dim3 const& big, dim3 const& tib, pfc::byte_t* dest, int const size, int const outer_idx, hipStream_t& stream) {
	mandel_kernel << <big, tib, 0, stream >> > (dest, size, outer_idx);
}

void call_mandel_kernel(dim3 const& big, dim3 const& tib, pfc::BGR_4_t* dest, int const size, int const outer_idx, hipStream_t& stream) {
	mandel_kernel << <big, tib, 0, stream >> > (dest, size, outer_idx);
}
